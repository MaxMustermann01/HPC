/*********************************************************************************
 * FILENAME         cuda_device_prop.cu
 *
 * DESCRIPTION      this program prints information about the used
 *                  graphic card to standard output
 *
 * AUTHORS          Christoph Klein
 *
 * LAST CHANGE      2 DEZ 2014
 *
 ********************************************************************************/

#include <stdlib.h>
#include <stdio.h>

int main (int argc, char *argv[]) {
    
    hipDeviceProp_t prop;
    int dev,count;
    hipGetDeviceCount(&count);
    hipGetDevice(&dev);
    hipGetDeviceProperties(&prop, dev); 

    printf("*** DEVICE PROPERTIES\n***\n");
    printf("***    DEVICE-COUNT %i\n", count);
    printf("***    ACTUAL-DEVICE %i\n", dev);
    printf("***    NAME %s\n", prop.name);
    printf("***    TOTAL-GLOBAL-MEM %.1f GB\n", prop.totalGlobalMem/1e9);
    printf("***    SHARED-MEM-PER-BLOCK %.1f kB\n", prop.sharedMemPerBlock/1e3);
    printf("***    MAX-THREADS-PER-BLOCK %i\n", prop.maxThreadsPerBlock);
    printf("***    MAX-THREADS-PER-MULTIPROCESSOR %i\n", prop.maxThreadsPerMultiProcessor);
    printf("***    MAX-THREADS-DIM (%i, %i, %i)\n"
            ,prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("***    MAX-GRID-DIM (%i, %i, %i)\n",
            prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("***    CLOCK-RATE %.2f GHz\n", prop.clockRate/1e6);
    printf("***    MULTI-PROC-COUNT %i\n", prop.multiProcessorCount);
    printf("***    MEMORY-CLOCK-RATE %.2f GHz\n", prop.memoryClockRate/1e6);
    printf("***    MEMORY-BUS-WIDTH %i bits\n", prop.memoryBusWidth);
    printf("***    L2-CACHE-SIZE %.1f kB\n", prop.l2CacheSize/1e3);
    printf("***    WARP-SIZE %i\n", prop.warpSize);
    printf("***    32BIT-REGISTERS-PER-BLOCK %i\n", prop.regsPerBlock);
    printf("***    ASYNCH-ENGINE-COUNT %i\n", prop.asyncEngineCount);
    printf("***    COMPUTE-CAPABILITY %i.%i\n", prop.major, prop.minor);
    printf("***\n");

}
