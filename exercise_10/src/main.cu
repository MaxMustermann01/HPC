#include "hip/hip_runtime.h"
/**************************************************************************************************
 *
 *       Computer Engineering Group, Heidelberg University - Introduction to HPC - sheet X
 *
 *                                 Group : Naumann, Schindler, Klein
 *
 *                                  File : main.cu
 *
 *                               Purpose : Matrix Multiplication
 *
 *************************************************************************************************/

#include <cmath>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <chCommandLine.h>
#include <chTimer.hpp>

#include "mmult_cpu.h"

const static int DEFAULT_MATRIX_WIDTH  = 1024;
const static int DEFAULT_BLOCK_DIM     =   27;

//
// Function Prototypes
//
void printHelp(char * /*programName*/);

//
// matMul_Kernel
//
__global__ void
matMul_Kernel(int matrixSize, float* matrixA, float* matrixB, float* matrixC)
{
    int elementIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int elementIdy = blockIdx.y * blockDim.y + threadIdx.y;
    
    int elementId = elementIdy * matrixSize + elementIdx;

    if (elementIdx < matrixSize && elementIdy < matrixSize) {
        matrixC[elementId] = 0; 
        for (int i = 0; i < matrixSize; i++) {
            matrixC[elementId] +=  matrixA[elementIdy * matrixSize + i]
                                * matrixB[matrixSize * i + elementIdx];
        }
    }
}

//
// Shared matMul_Kernel
//
__global__ void
shMatMul_Kernel(int matrixSize, float* matrixA, float* matrixB, float* matrixC)
{
    // shared memory //
    extern __shared__ float sh_Mem[];
    float *sh_MatrixA = &(sh_Mem[0]);
    // blockDim.x == blockDim.y is mandatory //
    float *sh_MatrixB = &(sh_Mem[blockDim.x * blockDim.y]);
    float *sh_MatrixC = &(sh_Mem[2*blockDim.x * blockDim.y]);
    
    // get indizes //
    int bbx = blockIdx.x, bby = blockIdx.y; 
    int tx = threadIdx.x, ty = threadIdx.y;
    int elementIdx = bbx * blockDim.x + tx;
    int elementIdy = bby * blockDim.y + ty;
    int id = ty * blockDim.x + tx; // thread id

    // actual indizes for elements which
    // should be loaded to shared memory
    // for matrix A and B
    int ax, ay, bx, by;
    
    // set output to zero first //
    sh_MatrixC[id] = 0;

    for (int i = 0; i < matrixSize; i += blockDim.x) {
        
        // each thread must load two matrix elements
        // into shared memory. ax and ay are the x and
        // y indizes of matrixA elements, which must be
        // loaded. bx and by analog for matrixB.
        ay = elementIdy;
        ax = i + tx;
        bx = elementIdx;
        by = i + ty;
        
        // this if else statements are mandatory if
        // matrixSize % blockDim.x != 0
        if (ax < matrixSize && ay < matrixSize)
            sh_MatrixA[id] = matrixA[ay * matrixSize + ax];
        else
            sh_MatrixA[id] = 0;
        if (bx < matrixSize && by < matrixSize)
            sh_MatrixB[id] = matrixB[by * matrixSize + bx];
        else 
            sh_MatrixB[id] = 0;

         __syncthreads();
        
         // add to result //
         for (int k = 0; k < blockDim.x; k++) {
             if (elementIdx < matrixSize && elementIdy < matrixSize) 
                 sh_MatrixC[id] += sh_MatrixA[ty * blockDim.y + k] 
                                * sh_MatrixB[ k * blockDim.y + tx];
         }
         __syncthreads();
    }
    
    // write to global memory //
    if (elementIdx < matrixSize && elementIdy < matrixSize) {
        matrixC[elementIdy * matrixSize + elementIdx] = sh_MatrixC[id];
    }
}
//
// Main
//
int
main(int argc, char * argv[])
{
    //
    // Show Help
    //
    bool showHelp = chCommandLineGetBool("h", argc, argv);
    if (!showHelp) {
        showHelp = chCommandLineGetBool("help", argc, argv);
    }

    if (showHelp) {
        printHelp(argv[0]);
        exit(0);
    }

    std::cout << "***" << std::endl
              << "*** Starting ..." << std::endl
              << "***" << std::endl;

    ChTimer memCpyH2DTimer, memCpyD2HTimer;
    ChTimer kernelTimer, cpuTimer;

    //
    // Allocate Memory
    //
    int matrixWidth = 0;
    chCommandLineGet<int>(&matrixWidth, "s", argc, argv);
    chCommandLineGet<int>(&matrixWidth, "size", argc, argv);
    matrixWidth = matrixWidth != 0 ? matrixWidth : DEFAULT_MATRIX_WIDTH;

    int matrixSize = matrixWidth * matrixWidth;

    //
    // Host Memory
    //
    bool pinnedMemory = chCommandLineGetBool("p", argc, argv);
    if (!pinnedMemory) {
        pinnedMemory = chCommandLineGetBool("pinned-memory",argc,argv);
    }

    float* h_matrixA = NULL;
    float* h_matrixB = NULL;
    float* h_matrixC = NULL;
    if (!pinnedMemory) {
        // Pageable
        h_matrixA = static_cast<float*>(malloc(
                        static_cast<size_t>(matrixSize * sizeof(*h_matrixA))));
        h_matrixB = static_cast<float*>(malloc(
                        static_cast<size_t>(matrixSize * sizeof(*h_matrixB))));
        h_matrixC = static_cast<float*>(calloc(
                        static_cast<size_t>(matrixSize), sizeof *h_matrixC));

    } else {
        // Pinned
        hipHostMalloc(&h_matrixA, static_cast<size_t>(matrixSize * sizeof(*h_matrixA)));
        hipHostMalloc(&h_matrixB, static_cast<size_t>(matrixSize * sizeof(*h_matrixB)));
        hipHostMalloc(&h_matrixC, static_cast<size_t>(matrixSize * sizeof(*h_matrixC)));
        memset ( h_matrixC, 0, matrixSize * sizeof(*h_matrixC) );
    }

    //
    // Device Memory
    //
    float* d_matrixA = NULL;
    float* d_matrixB = NULL;
    float* d_matrixC = NULL;
    hipMalloc(&d_matrixA, static_cast<size_t>(matrixSize * sizeof(*d_matrixA)));
    hipMalloc(&d_matrixB, static_cast<size_t>(matrixSize * sizeof(*d_matrixB)));
    hipMalloc(&d_matrixC, static_cast<size_t>(matrixSize * sizeof(*d_matrixC)));

    //
    // Check Pointers
    //
    if (h_matrixA == NULL || h_matrixB == NULL || h_matrixC == NULL ||
        d_matrixA == NULL || d_matrixB == NULL || d_matrixC == NULL )
    {
        std::cout << "\033[31m***" << std::endl
                  << "*** Error - Allocation of Memory failed!!!" << std::endl
                  << "***\033[0m" << std::endl;
        exit(-1);
    }

    //
    // Init Matrices
    //
    for (int i = 0; i < matrixSize; i++) {
        int x = i % matrixWidth;
        int y = i / matrixWidth;
        h_matrixA[i] = static_cast<float>(x * y);
        h_matrixB[i] = static_cast<float>(x + y);
    }

    //
    // Copy Data to the Device
    //
    memCpyH2DTimer.start();

    hipMemcpy(d_matrixA, h_matrixA, static_cast<size_t>(matrixSize * sizeof(*d_matrixA)), 
            hipMemcpyHostToDevice);
    hipMemcpy(d_matrixB, h_matrixB, static_cast<size_t>(matrixSize * sizeof(*d_matrixB)), 
            hipMemcpyHostToDevice);

    memCpyH2DTimer.stop();

    //
    // Get Kernel Launch Parameters
    //
    int blockSize = 0,
        gridSize = 0;

    // Block Dimension / Threads per Block
    chCommandLineGet<int>(&blockSize,"t", argc, argv);
    chCommandLineGet<int>(&blockSize,"threads-per-block", argc, argv);
    blockSize = blockSize != 0 ? blockSize : DEFAULT_BLOCK_DIM;

    if (blockSize > 32) {
        std::cout << "\033[31m***" << std::endl
                  << "*** Error - The number of threads per block is too big" << std::endl
                  << "***\033[0m" << std::endl;
        exit(-1);
    }

    gridSize = ceil(static_cast<float>(matrixWidth) / static_cast<float>(blockSize));

    dim3 grid_dim = dim3(gridSize, gridSize, 1);
    dim3 block_dim = dim3(blockSize, blockSize, 1);

    std::cout << "***" << std::endl
              << "*** Grid Dim:  " << grid_dim.x << "x" << grid_dim.y << "x" << grid_dim.z 
                      << std::endl
              << "*** Block Dim: " << block_dim.x << "x" << block_dim.y << "x" << block_dim.z 
                      << std::endl
              << "***" << std::endl;

    //
    // Calculate Shared Memory Size
    //
    int sharedMemSize = (blockSize * blockSize * 3) * sizeof(float);

    // get shared memory maximum size
    hipDeviceProp_t prop; 
    int devNr;
    hipGetDevice(&devNr);
    hipGetDeviceProperties(&prop, devNr);
    if (sharedMemSize > prop.sharedMemPerBlock 
            && chCommandLineGetBool("shared", argc, argv)) {
        std::cout << "\033[31m***" << std::endl
                  << "*** Error - MatrixSize is too big for shared memory" << std::endl
                  << "***\033[0m" << std::endl;
        exit(-1);
    }

    kernelTimer.start();

    //
    // Launch Kernel
    //
    if (!chCommandLineGetBool("shared", argc, argv)) {
        std::cout << "*** Execute MatMul_Kernel" << std::endl;
        matMul_Kernel <<< grid_dim, block_dim >>> (matrixWidth, d_matrixA, d_matrixB, d_matrixC);
    } else {
        std::cout << "*** Execute shMatMul_Kernel" << std::endl;
        shMatMul_Kernel<<<grid_dim, block_dim, sharedMemSize>>>(matrixWidth, d_matrixA, d_matrixB, d_matrixC);
    }

    //
    // Synchronize
    //
    hipDeviceSynchronize();

    //
    // Check for Errors
    //
    hipError_t hipError_t = hipGetLastError();
    if ( hipError_t != hipSuccess ) {
        std::cout << "\033[31m***" << std::endl
                  << "***ERROR*** " << hipError_t << " - " << hipGetErrorString(hipError_t)
                    << std::endl
                  << "***\033[0m" << std::endl;

        return -1;
    }

    kernelTimer.stop();

    //
    // Copy Back Data
    //
    memCpyD2HTimer.start();

    hipMemcpy(h_matrixC, d_matrixC, static_cast<size_t>(matrixSize * sizeof(*d_matrixC)), 
            hipMemcpyDeviceToHost);

    memCpyD2HTimer.stop();

    //
    // Check Result
    //
    bool dontCheckResult = chCommandLineGetBool("c", argc, argv);
    if (!dontCheckResult) {
        dontCheckResult = chCommandLineGetBool("no-check", argc, argv);
    }

    if (!dontCheckResult) {
        std::cout << "*** Execute CPU Matrix Multiplication" << std::endl;
        float* h_matrixD = static_cast<float*>(
                calloc(static_cast<size_t>(matrixSize), sizeof(*h_matrixD)));
        cpuTimer.start();
        MatrixMulOnHostBlocked(h_matrixA, h_matrixB, h_matrixD, 
                static_cast<long>(matrixWidth), 32);
        cpuTimer.stop();
        
        std::cout << "*** Compare the Results" << std::endl;
        bool resultOk = MatrixCompare(h_matrixC, h_matrixD, 
                static_cast<long>(matrixWidth));

        if (!resultOk) {
            std::cout << "\033[31m***" << std::endl
                      << "*** Error - The two matrices are different!!!" << std::endl
                      << "***\033[0m" << std::endl;

            exit(-1);
        }

        free(h_matrixD);
    }

    //
    // Print Meassurement Results
    //
    double totTime = memCpyH2DTimer.getTime()
                   + memCpyD2HTimer.getTime()
                   + kernelTimer.getTime();

    std::cout << "***" << std::endl
              << "*** Results:" << std::endl
              << "***    matrix-size " << matrixSize << std::endl
              << "***    time-to-copy-to-device " << 1e3 * memCpyH2DTimer.getTime()
                << " ms" << std::endl
              << "***    copy-bandwidth-to-device " 
                << 1e-9 * memCpyH2DTimer.getBandwidth(2 * matrixSize * sizeof(*h_matrixA))
                << " GB/s" << std::endl
              << "***    time-to-copy-from-device " << 1e3 * memCpyD2HTimer.getTime()
                << " ms" << std::endl
              << "***    copy-bandwidth-from-device " 
                << 1e-9 * memCpyD2HTimer.getBandwidth(matrixSize * sizeof(*h_matrixA))
                << " GB/s" << std::endl
              << "***    time-for-matrix-multiplication " << 1e3 * kernelTimer.getTime()
                  << " ms" << std::endl;
              if (!dontCheckResult) {
                  std::cout << "***    time-for-cpu-matrix-multiplication " 
                                << 1e3 * cpuTimer.getTime()
                                << " ms" << std::endl
                            << "***    speedup-kernel " 
                                << cpuTimer.getTime()/kernelTimer.getTime()
                                << std::endl
                            << "***    speedup-overall "
                                << cpuTimer.getTime()/totTime << std::endl;
              }
              std::cout << "***" << std::endl;

    if (chCommandLineGetBool("print-matrix", argc, argv) 
       && matrixWidth <= 16) {
        printOutMatrix(h_matrixC, matrixWidth);
    }

    // Free Memory
    if (!pinnedMemory) {
        free(h_matrixA);
        free(h_matrixB);
        free(h_matrixC);
    } else {
        hipHostFree(h_matrixA);
        hipHostFree(h_matrixB);
        hipHostFree(h_matrixC);
    }
    hipFree(d_matrixA);
    hipFree(d_matrixB);
    hipFree(d_matrixC);

    return 0;
}

void
printHelp(char * programName)
{
    std::cout << "Help:" << std::endl
              << "  Usage: " << std::endl
              << "  " << programName << " [-p] [-s <matrix_size>] [-t <threads_per_block>]" 
                << std::endl
              << "                 [-g <blocks_per_grid] [-c] [--print-matrix]" 
                << std::endl
              << "" << std::endl
              << "  -p|--pinned-memory" << std::endl
              << "  Use pinned Memory instead of pageable memory" << std::endl
              << "" << std::endl
              << "  -s <matrix_size>|--size <matix_size>" << std::endl
              << "  The width of the Matrix" << std::endl
              << "" << std::endl
              << "  -t <threads_per_block>|--threads-per-block <threads_per_block>" 
                << std::endl
              << "  The number of threads per block" << std::endl
              << "" << std::endl
              << "  -c|--no-checking" << std::endl
              << "  Do not check the result of the matrix multiplication" << std::endl
              << "" << std::endl
              << "  --print-matrix" << std::endl
              << "  Print the output matrix (only recommended for small matrices)" << std::endl << std::endl
              << "Further Information:" << std::endl
              << "  sizeof(int)   = " << sizeof(int) << " bytes" << std::endl
              << "  sizeof(float) = " << sizeof(float) << " bytes" << std::endl
              << std::endl;
}
